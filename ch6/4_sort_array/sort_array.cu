#include <iostream>
#include <vector>
#include <chrono>
#include <algorithm>
#include <cfloat>
#include <iomanip>
#include <hip/hip_runtime.h>

__global__ void oddEvenSortStepKernel(double *arr, int size, bool *swapped, bool isOddPhase) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int i = isOddPhase ? 2 * idx + 1 : 2 * idx;

    if (i < size - 1) {
        if (arr[i] > arr[i + 1]) {
            double temp = arr[i];
            arr[i] = arr[i + 1];
            arr[i + 1] = temp;
            *swapped = true;
        }
    }
}

void oddEvenSortGpu(double *arr, int size) {
    double *d_arr;
    bool *d_swapped;
    bool h_swapped;
    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    hipMalloc((void **)&d_arr, size * sizeof(double));
    hipMalloc((void **)&d_swapped, sizeof(bool));

    hipMemcpy(d_arr, arr, size * sizeof(double), hipMemcpyHostToDevice);

    do {
        h_swapped = false;
        hipMemcpy(d_swapped, &h_swapped, sizeof(bool), hipMemcpyHostToDevice);

        // Odd phase
        oddEvenSortStepKernel<<<blocks, threads>>>(d_arr, size, d_swapped, true);
        hipDeviceSynchronize();

        // Even phase
        oddEvenSortStepKernel<<<blocks, threads>>>(d_arr, size, d_swapped, false);
        hipDeviceSynchronize();

        // Check if any swaps occurred
        hipMemcpy(&h_swapped, d_swapped, sizeof(bool), hipMemcpyDeviceToHost);

    } while (h_swapped);

    hipMemcpy(arr, d_arr, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_arr);
    hipFree(d_swapped);
}

void oddEvenSortCpu(double *arr, int size) {
    bool swapped;

    do {
        swapped = false;

        // Odd phase
        for (int i = 1; i < size - 1; i += 2) {
            if (arr[i] > arr[i + 1]) {
                std::swap(arr[i], arr[i + 1]);
                swapped = true;
            }
        }

        // Even phase
        for (int i = 0; i < size - 1; i += 2) {
            if (arr[i] > arr[i + 1]) {
                std::swap(arr[i], arr[i + 1]);
                swapped = true;
            }
        }
    } while (swapped);
}

int main() {
    srand(static_cast<unsigned int>(time(0)));

    int n = 100'000;
    double *h_data = (double*)malloc(n * sizeof(double));
    double *h_data_gpu = (double*)malloc(n * sizeof(double));

    for (int i = 0; i < n; i++) {
        h_data[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    double* d_data;
    hipMalloc(&d_data, n * sizeof(double));
    hipMemcpy(d_data, h_data, n * sizeof(double), hipMemcpyHostToDevice);

    hipEvent_t startGpu, stopGpu;
    hipEventCreate(&startGpu);
    hipEventCreate(&stopGpu);

    hipEventRecord(startGpu);

    oddEvenSortGpu(d_data, n);

    hipEventRecord(stopGpu);

    hipEventSynchronize(stopGpu);
    float gpuDuration;
    hipEventElapsedTime(&gpuDuration, startGpu, stopGpu);
    std::cout << "GPU sorting time: " << gpuDuration << " ms" << std::endl;

    hipMemcpy(h_data_gpu, d_data, n * sizeof(double), hipMemcpyDeviceToHost);


    auto start = std::chrono::high_resolution_clock::now();
    oddEvenSortCpu(h_data, n);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpuDuration = end - start;
    std::cout << "CPU sorting time: " << cpuDuration.count() << " ms" << std::endl;

    double max_difference = 0;
    for (int i = 0; i < n; i++) {
        max_difference = std::max(max_difference, std::abs(h_data[i] - h_data_gpu[i]));
    }
    std::cout << "Max difference between CPU and GPU results: " << max_difference << std::endl;


    free(h_data);
    free(h_data_gpu);
    hipFree(d_data);
    hipEventDestroy(startGpu);
    hipEventDestroy(stopGpu);

    return 0;
}
