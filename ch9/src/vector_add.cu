#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../include/vector_add.h"

__global__ void vectorAddKernel(int *a, int *b, int *c, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {  
        c[idx] = a[idx] * b[idx];
    }
}

void vectorAdd(int *a, int *b, int *c, int N) {
    int *d_a;
    int *d_b;
    int *d_c;
    
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}