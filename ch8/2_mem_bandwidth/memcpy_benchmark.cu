#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE_MB(x) (long(x) * 1024 * 1024)  // Convert MB to bytes

void measureMemcpyBandwidth(long dataSize) {
    float *h_data, *d_data;
    hipHostMalloc(&h_data, dataSize, hipHostMallocDefault);
    hipMalloc(&d_data, dataSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpyAsync(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float bandwidth = (dataSize / (milliseconds / 1000.0)) / (1024.0 * 1024.0 * 1024.0);

    std::cout << "Data Size: " << dataSize / (1024 * 1024) << " MB, Time: " << milliseconds << " ms, Bandwidth: " << bandwidth << " GB/s\n";

    hipHostFree(h_data);
    hipFree(d_data);
}

int main() {
    std::cout << "Testing different transfer sizes:\n";
    measureMemcpyBandwidth(SIZE_MB(1));
    measureMemcpyBandwidth(SIZE_MB(10));
    measureMemcpyBandwidth(SIZE_MB(20));
    measureMemcpyBandwidth(SIZE_MB(30));
    measureMemcpyBandwidth(SIZE_MB(32));
    measureMemcpyBandwidth(SIZE_MB(40));
    measureMemcpyBandwidth(SIZE_MB(50));
    measureMemcpyBandwidth(SIZE_MB(100));
    measureMemcpyBandwidth(SIZE_MB(200));
    measureMemcpyBandwidth(SIZE_MB(300));
    measureMemcpyBandwidth(SIZE_MB(400));
    measureMemcpyBandwidth(SIZE_MB(500));
    measureMemcpyBandwidth(SIZE_MB(600));
    measureMemcpyBandwidth(SIZE_MB(700));
    measureMemcpyBandwidth(SIZE_MB(800));
    measureMemcpyBandwidth(SIZE_MB(900));
    measureMemcpyBandwidth(SIZE_MB(1024)); // 1GB

    return 0;
}
